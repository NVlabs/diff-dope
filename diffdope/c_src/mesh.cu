#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "common.h"
#include "mesh.h"


//------------------------------------------------------------------------
// Kernels

__global__ void xfmPointsFwdKernel(XfmKernelParams p)
{
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int pz = blockIdx.z * blockDim.z + threadIdx.z;

    __shared__ float mtx[4][4];
    if (threadIdx.x < 16)
        mtx[threadIdx.x % 4][threadIdx.x / 4] = p.matrix.fetch(p.matrix.nhwcIndex(pz, threadIdx.x / 4, threadIdx.x % 4, 0));
    __syncthreads();

    if (px >= p.gridSize.x)
        return;

    vec3f pos(
        p.points.fetch(p.points.nhwcIndex(pz, px, 0, 0)),
        p.points.fetch(p.points.nhwcIndex(pz, px, 1, 0)),
        p.points.fetch(p.points.nhwcIndex(pz, px, 2, 0))
    );

    if (p.isPoints)
    {
        p.out.store(p.out.nhwcIndex(pz, px, 0, 0), pos.x * mtx[0][0] + pos.y * mtx[1][0] + pos.z * mtx[2][0] + mtx[3][0]);
        p.out.store(p.out.nhwcIndex(pz, px, 1, 0), pos.x * mtx[0][1] + pos.y * mtx[1][1] + pos.z * mtx[2][1] + mtx[3][1]);
        p.out.store(p.out.nhwcIndex(pz, px, 2, 0), pos.x * mtx[0][2] + pos.y * mtx[1][2] + pos.z * mtx[2][2] + mtx[3][2]);
        p.out.store(p.out.nhwcIndex(pz, px, 3, 0), pos.x * mtx[0][3] + pos.y * mtx[1][3] + pos.z * mtx[2][3] + mtx[3][3]);
    }
    else
    {
        p.out.store(p.out.nhwcIndex(pz, px, 0, 0), pos.x * mtx[0][0] + pos.y * mtx[1][0] + pos.z * mtx[2][0]);
        p.out.store(p.out.nhwcIndex(pz, px, 1, 0), pos.x * mtx[0][1] + pos.y * mtx[1][1] + pos.z * mtx[2][1]);
        p.out.store(p.out.nhwcIndex(pz, px, 2, 0), pos.x * mtx[0][2] + pos.y * mtx[1][2] + pos.z * mtx[2][2]);
    }
}

__global__ void xfmPointsBwdKernel(XfmKernelParams p)
{
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int pz = blockIdx.z * blockDim.z + threadIdx.z;

    __shared__ float mtx[4][4];
    if (threadIdx.x < 16)
        mtx[threadIdx.x % 4][threadIdx.x / 4] = p.matrix.fetch(p.matrix.nhwcIndex(pz, threadIdx.x / 4, threadIdx.x % 4, 0));
    __syncthreads();

    if (px >= p.gridSize.x)
        return;

    vec3f pos(
        p.points.fetch(p.points.nhwcIndex(pz, px, 0, 0)),
        p.points.fetch(p.points.nhwcIndex(pz, px, 1, 0)),
        p.points.fetch(p.points.nhwcIndex(pz, px, 2, 0))
    );

    vec4f d_out(
        p.out.fetch(p.out.nhwcIndex(pz, px, 0, 0)),
        p.out.fetch(p.out.nhwcIndex(pz, px, 1, 0)),
        p.out.fetch(p.out.nhwcIndex(pz, px, 2, 0)),
        p.out.fetch(p.out.nhwcIndex(pz, px, 3, 0))
    );

    if (p.isPoints)
    {
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 0, 0), d_out.x * mtx[0][0] + d_out.y * mtx[0][1] + d_out.z * mtx[0][2] + d_out.w * mtx[0][3]);
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 1, 0), d_out.x * mtx[1][0] + d_out.y * mtx[1][1] + d_out.z * mtx[1][2] + d_out.w * mtx[1][3]);
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 2, 0), d_out.x * mtx[2][0] + d_out.y * mtx[2][1] + d_out.z * mtx[2][2] + d_out.w * mtx[2][3]);
    }
    else
    {
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 0, 0), d_out.x * mtx[0][0] + d_out.y * mtx[0][1] + d_out.z * mtx[0][2]);
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 1, 0), d_out.x * mtx[1][0] + d_out.y * mtx[1][1] + d_out.z * mtx[1][2]);
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 2, 0), d_out.x * mtx[2][0] + d_out.y * mtx[2][1] + d_out.z * mtx[2][2]);
    }
}

__global__ void xfmPointsBwdFullKernel(XfmKernelParams p)
{
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int pz = blockIdx.z * blockDim.z + threadIdx.z;

    __shared__ float mtx[4][4];
    if (threadIdx.x < 16)
        mtx[threadIdx.x % 4][threadIdx.x / 4] = p.matrix.fetch(p.matrix.nhwcIndex(pz, threadIdx.x / 4, threadIdx.x % 4, 0));
    __syncthreads();

    if (px >= p.gridSize.x)
        return;

    vec3f pos(
        p.points.fetch(p.points.nhwcIndex(pz, px, 0, 0)),
        p.points.fetch(p.points.nhwcIndex(pz, px, 1, 0)),
        p.points.fetch(p.points.nhwcIndex(pz, px, 2, 0))
    );

    vec4f d_out(
        p.out.fetch(p.out.nhwcIndex(pz, px, 0, 0)),
        p.out.fetch(p.out.nhwcIndex(pz, px, 1, 0)),
        p.out.fetch(p.out.nhwcIndex(pz, px, 2, 0)),
        p.out.fetch(p.out.nhwcIndex(pz, px, 3, 0))
    );

    if (p.isPoints)
    {
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 0, 0), d_out.x * mtx[0][0] + d_out.y * mtx[0][1] + d_out.z * mtx[0][2] + d_out.w * mtx[0][3]);
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 1, 0), d_out.x * mtx[1][0] + d_out.y * mtx[1][1] + d_out.z * mtx[1][2] + d_out.w * mtx[1][3]);
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 2, 0), d_out.x * mtx[2][0] + d_out.y * mtx[2][1] + d_out.z * mtx[2][2] + d_out.w * mtx[2][3]);
    }
    else
    {
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 0, 0), d_out.x * mtx[0][0] + d_out.y * mtx[0][1] + d_out.z * mtx[0][2]);
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 1, 0), d_out.x * mtx[1][0] + d_out.y * mtx[1][1] + d_out.z * mtx[1][2]);
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 2, 0), d_out.x * mtx[2][0] + d_out.y * mtx[2][1] + d_out.z * mtx[2][2]);
    }

    unsigned int id = threadIdx.x % p.padFactor;

    // Gradient to matrix weights
    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 0, 0, id), pos.x * d_out.x);
    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 0, 1, id), pos.y * d_out.x);
    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 0, 2, id), pos.z * d_out.x);
    if (p.isPoints)
        atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 0, 3, id), d_out.x);

    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 1, 0, id), pos.x * d_out.y);
    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 1, 1, id), pos.y * d_out.y);
    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 1, 2, id), pos.z * d_out.y);
    if (p.isPoints)
        atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 1, 3, id), d_out.y);

    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 2, 0, id), pos.x * d_out.z);
    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 2, 1, id), pos.y * d_out.z);
    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 2, 2, id), pos.z * d_out.z);
    if (p.isPoints)
        atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 2, 3, id), d_out.z);

    if (p.isPoints)
    {
        atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 3, 0, id), pos.x * d_out.w);
        atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 3, 1, id), pos.y * d_out.w);
        atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 3, 2, id), pos.z * d_out.w);
        atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 3, 3, id), d_out.w);
    }
}

__global__ void xfmPointsBwdMtxKernel(XfmKernelParams p)
{
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int pz = blockIdx.z * blockDim.z + threadIdx.z;

    if (px >= p.gridSize.x)
        return;

    vec3f pos(
        p.points.fetch(p.points.nhwcIndex(pz, px, 0, 0)),
        p.points.fetch(p.points.nhwcIndex(pz, px, 1, 0)),
        p.points.fetch(p.points.nhwcIndex(pz, px, 2, 0))
    );

    vec4f d_out(
        p.out.fetch(p.out.nhwcIndex(pz, px, 0, 0)),
        p.out.fetch(p.out.nhwcIndex(pz, px, 1, 0)),
        p.out.fetch(p.out.nhwcIndex(pz, px, 2, 0)),
        p.out.fetch(p.out.nhwcIndex(pz, px, 3, 0))
    );

    unsigned int id = threadIdx.x % p.padFactor;

    // Gradient to matrix weights
    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 0, 0, id), pos.x * d_out.x);
    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 0, 1, id), pos.y * d_out.x);
    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 0, 2, id), pos.z * d_out.x);
    if (p.isPoints)
        atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 0, 3, id), d_out.x);

    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 1, 0, id), pos.x * d_out.y);
    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 1, 1, id), pos.y * d_out.y);
    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 1, 2, id), pos.z * d_out.y);
    if (p.isPoints)
        atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 1, 3, id), d_out.y);

    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 2, 0, id), pos.x * d_out.z);
    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 2, 1, id), pos.y * d_out.z);
    atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 2, 2, id), pos.z * d_out.z);
    if (p.isPoints)
        atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 2, 3, id), d_out.z);

    if (p.isPoints)
    {
        atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 3, 0, id), pos.x * d_out.w);
        atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 3, 1, id), pos.y * d_out.w);
        atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 3, 2, id), pos.z * d_out.w);
        atomicAdd((float*)p.matrix.d_val + p.matrix.nhwcIndexContinuous(pz, 3, 3, id), d_out.w);
    }
}
